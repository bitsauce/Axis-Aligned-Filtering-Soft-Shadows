#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Calculates the disparity between the two input buffers
// This program assumes that the input buffers
// are in the range [0, 1]
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float3, 2> input_buffer_0;
rtBuffer<float3, 2> input_buffer_1;
rtBuffer<float3, 2> difference_buffer;

RT_PROGRAM void calculate_difference()
{
	const float3 diff = input_buffer_0[launch_index] - input_buffer_1[launch_index];
	difference_buffer[launch_index] = make_float3(dot(diff, diff) / 3.f * 20.f);
}
