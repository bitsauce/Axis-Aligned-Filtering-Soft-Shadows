#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Calculates the disparity between the two input buffers
// This program assumes that the input buffers
// are in the range [0, 1]
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float3, 2> input_buffer_0;
rtBuffer<float3, 2> input_buffer_1;
rtBuffer<float3, 2> disparity_buffer;

RT_PROGRAM void calculate_disparity()
{
	disparity_buffer[launch_index] = make_float3(length(input_buffer_0[launch_index] - input_buffer_1[launch_index]) / 4.f);
}
