#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Gaussian blur - using input_buffer.w as filter size
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2> main_output;
rtBuffer<float4, 2> blur_output;

float gauss(const float x, const float y, const float std)
{
	return expf(-(x * x + y * y) / (2.f * std * std)) / (2.f * M_PIf * std * std);
}

RT_PROGRAM void blur()
{
	size_t2 screen = main_output.size();
	const int kernel_size = int(main_output[launch_index].w) / 2;

	float3 color = make_float3(0.f);
	float weight = 0.f;
	uint2 pixel_index;
	for(int y = -kernel_size; y <= kernel_size; y++)
	{
		pixel_index.y = launch_index.y + y; // Expoiting interger underflow
		if(pixel_index.y >= screen.y) continue;
		for(int x = -kernel_size; x <= kernel_size; x++)
		{
			pixel_index.x = launch_index.x + x;
			if(pixel_index.x >= screen.x) continue;

			const float g = gauss(x, y, 1.f);
			color += make_float3(main_output[pixel_index]) * g;
			weight += g;
		}
	}

	blur_output[launch_index] = make_float4(color / weight, 1.f);
}
