#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Gaussian blur - using input_buffer.w as filter size
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2> diffuse_buffer;
rtBuffer<float, 2> beta_buffer;
rtBuffer<float2, 2> projected_distances_buffer;
rtBuffer<float4, 2> blur_h_buffer;
rtBuffer<float4, 2> blur_v_buffer;

float gauss1D(const float x, const float std)
{
	const float sqrt_2_pi = sqrtf(2.f * M_PIf);
	return expf(-(x * x) / (2.f * std * std)) / (sqrt_2_pi * std);
}

RT_PROGRAM void blurH()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];
	const int kernel_size = 25.f * beta / 16.0f; // TODO: Experiment with different kernel_sizes -- kernel as a function of beta?

	if(beta == 0.f) {
		blur_h_buffer[launch_index] = make_float4(make_float3(diffuse_buffer[launch_index]), 1.f);
		return;
	}

	float3 color = make_float3(0.f);
	float sum = 0.f;
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		const uint x = launch_index.x + i;
		if(x >= screen.x) continue; // Explointing interger underflow for x < 0

		float2 center = projected_distances_buffer[launch_index];
		float2 p = projected_distances_buffer[make_uint2(x, launch_index.y)];
		const float offset = length(center - p);

		const float w = gauss1D(offset/300.f, beta);
		color += make_float3(diffuse_buffer[make_uint2(x, launch_index.y)]) * w;
		sum += w;
	}

	blur_h_buffer[launch_index] = make_float4(color / sum, 1.f);
}

RT_PROGRAM void blurV()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];
	const int kernel_size = 25.f * beta / 16.0f;

	if(beta == 0.f) {
		blur_v_buffer[launch_index] = make_float4(make_float3(diffuse_buffer[launch_index]), 1.f);
		return;
	}

	float3 color = make_float3(0.f);
	float sum = 0.f;
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		const uint y = launch_index.y + i;
		if(y >= screen.y) continue;
		const float w = gauss1D(i, beta);
		color += make_float3(blur_h_buffer[make_uint2(launch_index.x, y)]) * w;
		sum += w;
	}

	blur_v_buffer[launch_index] = make_float4(color / sum, 1.f);
}
