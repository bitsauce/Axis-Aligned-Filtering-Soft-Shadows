#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Gaussian blur - using input_buffer.w as filter size
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float3, 2> diffuse_buffer;
rtBuffer<float,  2> beta_buffer;
rtBuffer<float,  2> object_id_buffer;
rtBuffer<float2, 2> projected_distances_buffer;
rtBuffer<float3, 2> blur_h_buffer;
rtBuffer<float3, 2> blur_v_buffer;
rtBuffer<float3, 2> geometry_normal_buffer;

float gauss1D(const float x, const float std)
{
	const float sqrt_2_pi = sqrtf(2.f * M_PIf);
	return expf(-(x * x) / (2.f * std * std)) / (sqrt_2_pi * std);
}

RT_PROGRAM void blurH()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];

	// TODO: Experiment with different kernel_sizes -- kernel as a function of beta?
	const int kernel_size = min(beta * 4.0f, 10.0f);

	if(beta == 0.f) {
		blur_h_buffer[launch_index] = diffuse_buffer[launch_index];
		return;
	}

	float object_id = object_id_buffer[launch_index];
	float3 geometry_normal = geometry_normal_buffer[launch_index];
	float3 color = make_float3(0.f);
	float sum = 0.f;
	float2 center = projected_distances_buffer[launch_index];
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		// Explointing interger underflow when pos.x < 0
		const uint2 pos = make_uint2(launch_index.x + i, launch_index.y);
		if(pos.x >= screen.x || object_id != object_id_buffer[pos]) continue;

		float2 p = projected_distances_buffer[pos];
		const float offset = length(center - p);

		const float w = gauss1D(offset, beta) * dot(geometry_normal, geometry_normal_buffer[pos]);
		color += diffuse_buffer[pos] * w;
		sum += w;
	}

	blur_h_buffer[launch_index] = color / sum;
}

RT_PROGRAM void blurV()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];
	const int kernel_size = min(beta * 4.0f, 10.0f);

	if(beta == 0.f) {
		blur_v_buffer[launch_index] = diffuse_buffer[launch_index];
		return;
	}
	
	float object_id = object_id_buffer[launch_index];
	float3 geometry_normal = geometry_normal_buffer[launch_index];
	float3 color = make_float3(0.f);
	float sum = 0.f;
	float2 center = projected_distances_buffer[launch_index];
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		const uint2 pos = make_uint2(launch_index.x, launch_index.y + i);
		if(pos.y >= screen.y || object_id != object_id_buffer[pos]) continue;

		float2 p = projected_distances_buffer[pos];
		const float offset = length(center - p);

		const float w = gauss1D(offset, beta) * dot(geometry_normal, geometry_normal_buffer[pos]);
		color += blur_h_buffer[pos] * w;
		sum += w;
	}

	blur_v_buffer[launch_index] = color / sum;
}
