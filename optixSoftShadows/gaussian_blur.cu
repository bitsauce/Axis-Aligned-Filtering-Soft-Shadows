#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Gaussian blur - using input_buffer.w as filter size
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float4, 2> diffuse_buffer;
rtBuffer<float, 2> beta_buffer;
rtBuffer<float4, 2> blur_output;

float gauss2D(const float x, const float y, const float std)
{
	return expf(-(x * x + y * y) / (2.f * std * std)) / (2.f * M_PIf * std * std);
}

float gauss1D(const float x, const float std)
{
	return expf(-(x * x) / (2.f * std * std)) / (2.f * M_PIf * std * std);
}

RT_PROGRAM void blurH()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];
	const int kernel_size = 25; // TODO: Experiment with different kernel_sizes -- kernel as a function of beta?

	if(beta == 0.f) {
		blur_output[launch_index] = make_float4(make_float3(diffuse_buffer[launch_index]), 1.f);
		return;
	}

	float3 color = make_float3(0.f);
	float sum = 0.f;
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		const uint x = launch_index.x + i;
		if(x >= screen.x) continue;
		const float w = gauss1D(x, beta);
		color += make_float3(diffuse_buffer[make_uint2(x, launch_index.y)]) * w;
		sum += w;
	}

	blur_output[launch_index] = make_float4(color / sum, 1.f);
}

RT_PROGRAM void blurV()
{
	size_t2 screen = diffuse_buffer.size();
	const float beta = beta_buffer[launch_index];
	const int kernel_size = 25;

	if(beta == 0.f) {
		blur_output[launch_index] = make_float4(make_float3(diffuse_buffer[launch_index]), 1.f);
		return;
	}

	float3 color = make_float3(0.f);
	float sum = 0.f;
	for(int i = -kernel_size; i <= kernel_size; i++)
	{
		const uint y = launch_index.y + i;
		if(y >= screen.y) continue;
		const float w = gauss1D(y, beta);
		color += make_float3(blur_output[make_uint2(launch_index.x, y)]) * w;
		sum += w;
	}

	blur_output[launch_index] = make_float4(color / sum, 1.f);
}
