#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "structs.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-3f
#define NUM_SAMPLES 4000

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4, 2> diffuse_buffer;             // Diffuse color buffer

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry, , );

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_ground_truth, prd_diffuse, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Light sources
rtBuffer<ParallelogramLight> lights;

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------

RT_PROGRAM void trace_ray()
{
	size_t2 screen = diffuse_buffer.size(); // Screen size
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // Pixel coordinate in [-1, 1]
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Create ray from camera into scene
	Ray ray(ray_origin, ray_direction, GROUND_TRUTH_RAY, EPSILON);

	// Per radiance data
	PerRayData_ground_truth prd;
	prd.seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0);//frame_number);

	// Trace geometry
	rtTrace(scene_geometry, ray, prd);

	// Set resulting diffuse color and beta
	diffuse_buffer[launch_index] = make_float4(prd.color, 1.f);
}

//-----------------------------------------------------------------------------
// Lambertian surface closest-hit
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint, object_id, , );

#define FLT_MAX 3.402823466e+38F

RT_PROGRAM void diffuse()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;
	
	unsigned int seed = prd_diffuse.seed;
	for(int i = 0; i < lights.size(); ++i)
	{
		ParallelogramLight light = lights[i];
		const float3 light_center = light.corner + light.v1 * 0.5f + light.v2 * 0.5f;


		// DEBUG: Show the light
		if(length(hit_point - light_center) < 10.0f)
		{
			color = make_float3(1.f, 1.f, 1.f);
		}
		else
		{
			const int num_samples = NUM_SAMPLES;
			const float avg_factor = 1.0f / float(NUM_SAMPLES);
			for(int j = 0; j < num_samples; j++)
			{
				// Choose random point on light
				const float z1 = rnd(seed);
				const float z2 = rnd(seed);
				const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

				// Sample color
				float3 L = normalize(light_pos - hit_point);
				float nDl = dot(ffnormal, L);
				if(nDl > 0.0f) // Check if light is behind
				{
					float Ldist = length(light_pos - hit_point);

					// Cast shadow ray
					PerRayData_shadow shadow_prd;
					shadow_prd.hit = false;

					Ray shadow_ray(hit_point, L, SHADOW_RAY, EPSILON, Ldist);
					rtTrace(scene_geometry, shadow_ray, shadow_prd);

					// Set color if we hit the light
					if(!shadow_prd.hit)
					{
						color += Kd * nDl * diffuse_color * avg_factor;
					}
				}
			}
		}
	}
	prd_diffuse.color = color;
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.hit_point = ray.origin + t_hit * ray.direction;
	prd_shadow.hit = true;
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_diffuse.color = bg_color;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	diffuse_buffer[launch_index] = make_float4(bad_color, 1.f);
}