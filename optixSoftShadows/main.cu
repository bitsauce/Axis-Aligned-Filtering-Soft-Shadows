#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

#define EPSILON  1.e-3f

struct PerRayData_pathtrace
{
	float3 result;
	float3 radiance;
	float3 attenuation;
	float3 origin;
	float3 direction;
	unsigned int seed;
	int depth;
	int countEmitted;
	int done;
};

// Input pixel-coordinate
// An uint2 value (x, y) bound to internal
// state variable, rtLaunchIndex
rtDeclareVariable(uint2, launch_index, rtLaunchIndex,);

// Output buffer (final image)
// A 2-dimensional buffer of float4s
rtBuffer<float4, 2> output_buffer;

// Simulation time variable passed from program
rtDeclareVariable(float, time,,);
rtDeclareVariable(rtObject, scene_geometry,,);

rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

// Main ray program
RT_PROGRAM void trace_ray()
{
	float intensity = fmodf(time, 2.0f);

	float3 ray_origin = make_float3(0.0f);
	float3 ray_direction = make_float3(1.0f, 0.0f, 0.0f);

	// Initialze per-ray data
	PerRayData_pathtrace prd;
	prd.result = make_float3(0.f);
	prd.attenuation = make_float3(1.f);
	prd.countEmitted = true;
	prd.done = false;
	prd.seed = 0;
	prd.depth = 0;

	Ray ray = make_Ray(ray_origin, ray_direction, 0, EPSILON, RT_DEFAULT_MAX);
	rtTrace(scene_geometry, ray, prd);

	output_buffer[launch_index] = make_float4(make_float3(prd.radiance), 0.f);
}


//
// Returns solid color for miss rays
//
rtDeclareVariable(float3, bg_color, , );
RT_PROGRAM void miss()
{
	prd.radiance = bg_color;
}


//
// Returns shading normal as the surface shading result
// 
RT_PROGRAM void closest_hit_radiance0()
{
	prd.radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal))*0.5f + 0.5f;
}


//
// Set pixel to solid color upon failur
//
RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_color(bad_color);
}