#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "structs.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-3f

//--------------------------------------------------------------
// Per-ray data structs
//--------------------------------------------------------------

struct PerRayData_diffuse
{
	float3       color;         // Diffuse color
	float        depth;			// Sample depth
	float2       projected_distance;
	float        object_id;
	float        beta;			// Filter width (screen-space standard deviation)
	float		 num_samples;	// Number of adaptive samples
	unsigned int seed;          // Seed for random sampling
};

struct PerRayData_shadow
{
	float3 attenuation;
	float3 hit_point;
};

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float4, 2> diffuse_buffer;             // Diffuse color buffer
rtBuffer<float,  2> beta_buffer;                // Beta buffer (gaussian standard deviation)
rtBuffer<float,  2> depth_buffer;               // Depth buffer
rtBuffer<float,  2> object_id_buffer;           // Object id buffer
rtBuffer<float,  2> num_samples_buffer;           // Sample number buffer
rtBuffer<float2, 2> projected_distances_buffer; // Projected distances buffer (offset of screen-space gaussian)

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry, , );

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_diffuse, prd_diffuse, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Light sources
rtBuffer<ParallelogramLight> lights;

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------

RT_PROGRAM void trace_ray()
{
	size_t2 screen = diffuse_buffer.size(); // Screen size
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // Pixel coordinate in [-1, 1]
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Create ray from camera into scene
	Ray ray(ray_origin, ray_direction, DIFFUSE_RAY, EPSILON);

	// Per radiance data
	PerRayData_diffuse prd;
	prd.seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0);//frame_number);
	prd.beta = 0.f;

	// Trace geometry
	rtTrace(scene_geometry, ray, prd);

	// Set resulting diffuse color and beta
	diffuse_buffer[launch_index] = make_float4(prd.color, 1.f);
	beta_buffer[launch_index] = prd.beta;
	depth_buffer[launch_index] = prd.depth;
	projected_distances_buffer[launch_index] = prd.projected_distance;
	object_id_buffer[launch_index] = prd.object_id;
	num_samples_buffer[launch_index] = prd.num_samples;
}

//-----------------------------------------------------------------------------
// Lambertian surface closest-hit
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint, object_id, , );

#define FLT_MAX 3.402823466e+38F

RT_PROGRAM void sample_distances(unsigned int& seed, ParallelogramLight light, float3 ffnormal, float3 hit_point, float& d2_min, float& d2_max)
{
	// Choose random point on light
	const float z1 = rnd(seed);
	const float z2 = rnd(seed);
	const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

	float3 L = normalize(light_pos - hit_point);
	float nDl = dot(ffnormal, L);
	//if(nDl > 0.0f) // Check if light is behind
	{
		// TODO: Maybe d1 should be average of these?
		float Ldist = length(light_pos - hit_point);

		// Cast shadow ray
		PerRayData_shadow shadow_prd;
		shadow_prd.attenuation = make_float3(1.0f);

		Ray shadow_ray(hit_point, L, SHADOW_RAY, EPSILON, Ldist);
		rtTrace(scene_geometry, shadow_ray, shadow_prd);

		float3 light_attenuation = shadow_prd.attenuation;
		if (fmaxf(light_attenuation) <= 0.0f) // If light source was occluded
		{
			const float d2 = length(shadow_prd.hit_point - light_pos);

			// Store min d2
			if (d2 < d2_min)
			{
				d2_min = d2;
			}

			// Store max d2
			if (d2 > d2_max)
			{
				d2_max = d2;
			}
		}
	}
}

RT_PROGRAM void diffuse()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;
	
	unsigned int seed = prd_diffuse.seed;
	for(int i = 0; i < lights.size(); ++i)
	{
		ParallelogramLight light = lights[i];
		const float3 light_center = light.corner + light.v1 * 0.5f + light.v2 * 0.5f;

		Matrix3x3 projection_matrix;
		projection_matrix.setRow(0, light.v1);
		projection_matrix.setRow(1, light.v2);
		projection_matrix.setRow(2, light.normal);

		float3 p_projected = projection_matrix * hit_point;
		prd_diffuse.projected_distance = make_float2(p_projected);


		// Sample color
		float3 L = normalize(light_center - hit_point);
		float nDl = dot(ffnormal, L);
		if (nDl > 0.0f) // Check if light is behind
		{
			float Ldist = length(light_center - hit_point);

			// Cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);

			Ray shadow_ray(hit_point, L, SHADOW_RAY, EPSILON, Ldist);
			rtTrace(scene_geometry, shadow_ray, shadow_prd);

			float3 light_attenuation = shadow_prd.attenuation;
			if (fmaxf(light_attenuation) > 0.0f) // If we hit the light
			{
				float3 Lc = light_attenuation * diffuse_color;
				color += Kd * nDl * Lc;

				// Apply specularity
				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if (nDh > 0)
				{
					color += Ks * Lc * pow(nDh, phong_exp);
				}
			}
		}

		// Send 9 rays
		float d2_min = FLT_MAX;  // Min distance from light to occluder
		float d2_max = -FLT_MAX; // Max distance from light to occluder
		float d1 = length(hit_point - light_center); // Distance from light to receiver
		for(int j = 0; j < 9; j++)
		{
			sample_distances(seed, light, ffnormal, hit_point, d2_min, d2_max);
		}

		// DEBUG: Show the light
		if(d1 < 10.0f)
		{
			color = make_float3(1.f, 1.f, 1.f);
		}

		// Constants from the paper
		const float k = 3.f;
		const float alpha = 1.f;
		const float mu = 2.f;
		const float max_num_samples = 50.f;

		// Standard deviation of Gaussian of the light
		// TODO: Experiment with different sigmas
		const float sigma = 130.f / 2.f;

		const float s1 = (d1 / d2_min) - 1.f;
		const float s2 = (d1 / d2_max) - 1.f;
		const float inv_s2 = alpha / (1.f + s2);

		const float depth = length(t_hit * ray.direction);
		const float omega_max_pix = 1.f / depth;
		// TODO: d should be calculated as the average 3D eucledean distance between this pixels hitpoint and it's neighbouring pixels
		// persumably this should be calculated in its own pass beforehand
		const float omega_max_x = inv_s2 * omega_max_pix;

		// If this pixel was occluded (that is, d2_max > 0)
		if(d2_max > 0.f)
		{
			// Calculate filter width at current pixel
			const float beta = 1.f / k * 1.f / mu * max(sigma * ((d1 / d2_max) - 1.f), //1.f / omega_max_x); // TODO: Calculate the omega_max_x and use it
																					   -1000.0f);
			prd_diffuse.beta = beta;
		}

		// Calculate pixel area and light area
		const float Ap = 1.f / (omega_max_pix * omega_max_pix);
		const float Al = 4.f * sigma * sigma;

		// Calcuate number of additional samples
		const float num_samples = 50.0f; // min(mu * 2 / s2 * sqrtf(Ap / Al), max_num_samples); //min(4.f * powf(1.f + mu * (s1 / s2), 2.f) * powf(mu * 2 / s2 * sqrtf(Ap / Al) + inv_s2, 2.f), max_num_samples);
		prd_diffuse.num_samples = num_samples;

		for (int j = 0; j < (int)num_samples; j++)
		{
			sample_distances(seed, light, ffnormal, hit_point, d2_min, d2_max);
		}
	}
	prd_diffuse.color = color;
	prd_diffuse.depth = length(hit_point - ray.origin);
	prd_diffuse.object_id = float(object_id);
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.hit_point = ray.origin + t_hit * ray.direction;
	prd_shadow.attenuation = make_float3(0);
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_diffuse.color = bg_color;
	prd_diffuse.depth = 0.f;
	prd_diffuse.object_id = 0.f;
	prd_diffuse.num_samples = 0.f;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	diffuse_buffer[launch_index] = make_float4(bad_color, 1.f);
	beta_buffer[launch_index] = 0.f;
	object_id_buffer[launch_index] = 0.f;
	num_samples_buffer[launch_index] = 0.f;
}