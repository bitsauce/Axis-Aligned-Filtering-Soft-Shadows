#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

using namespace optix;

#define EPSILON  1.e-3f

//--------------------------------------------------------------
// Per ray data struct
//--------------------------------------------------------------

struct PerRayData_radiance
{
	float3 result;
};

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
// An uint2 value (x, y) bound to internal state variable "rtLaunchIndex"
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// Output buffer (final image)
// A 2-dimensional buffer of float4s
rtBuffer<float4, 2> output_buffer;

// Shading normal from intersection program
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry,,);

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U, , );
rtDeclareVariable(float3, V, , );
rtDeclareVariable(float3, W, , );

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------
RT_PROGRAM void trace_ray()
{
	size_t2 screen = output_buffer.size();
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // pixel-coordinate [-1, 1] range
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Initialze per-ray data
	PerRayData_radiance prd;

	// Trace the ray in the direction of the camera
	Ray ray = make_Ray(ray_origin, ray_direction, 0, EPSILON, RT_DEFAULT_MAX);
	rtTrace(scene_geometry, ray, prd);

	// Set output color
	output_buffer[launch_index] = make_float4(prd.result, 0.f);
}

//--------------------------------------------------------------
// Closest hit radiance
//--------------------------------------------------------------
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );

RT_PROGRAM void closest_hit_radiance()
{
	prd_radiance.result = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------
rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------
rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}