#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "structs.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-3f

//--------------------------------------------------------------
// Per-ray data structs
//--------------------------------------------------------------

struct PerRayData_diffuse
{
	float3       result;
	unsigned int seed;
};

struct PerRayData_shadow
{
	float3 attenuation;
};

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
// An uint2 value (x, y) bound to internal state variable "rtLaunchIndex"
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// Output buffer (final image)
// A 2-dimensional buffer of float4s
rtBuffer<float4, 2> output_buffer;

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry,,);

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_diffuse, prd_diffuse, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Light sources
rtBuffer<ParallelogramLight> lights;

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------

RT_PROGRAM void trace_ray()
{
	size_t2 screen = output_buffer.size(); // Screen size
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // Pixel coordinate in [-1, 1]
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Create ray from camera into scene
	Ray ray(ray_origin, ray_direction, 0, EPSILON);

	// Per radiance data
	PerRayData_diffuse prd;
	prd.seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0);//frame_number);

	// Trace geometry
	rtTrace(scene_geometry, ray, prd);

	// Set resulting color
	output_buffer[launch_index] = make_float4(prd.result, 1.0f);
}

//-----------------------------------------------------------------------------
// Lambertian surface closest-hit
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void diffuse()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	unsigned int seed = prd_diffuse.seed;
	for(int i = 0; i < lights.size(); ++i)
	{
		//BasicLight light = lights[i];


		// Choose random point on light
		ParallelogramLight light = lights[i];
		const float z1 = rnd(seed);
		const float z2 = rnd(seed);
		const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;


		float3 L = normalize(light_pos - hit_point);
		float nDl = dot(ffnormal, L);

		if(nDl > 0.0f)
		{
			// Cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light_pos - hit_point);
			Ray shadow_ray(hit_point, L, 1, EPSILON, Ldist);
			rtTrace(scene_geometry, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if(fmaxf(light_attenuation) > 0.0f)
			{
				float3 Lc = light_attenuation /* * light_color */;
				color += Kd * nDl * Lc;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if(nDh > 0)
					color += Ks * Lc * pow(nDh, phong_exp);
			}

		}
	}
	prd_diffuse.result = color;
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.attenuation = make_float3(0);
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_diffuse.result = bg_color;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}