#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include "structs.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-3f

//--------------------------------------------------------------
// Per-ray data structs
//--------------------------------------------------------------

struct PerRayData_diffuse
{
	float3       result;
	unsigned int seed;
};

struct PerRayData_shadow
{
	float3 attenuation;
	float3 hit_point;
};

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
// An uint2 value (x, y) bound to internal state variable "rtLaunchIndex"
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// Output buffer (final image)
// A 2-dimensional buffer of float4s
rtBuffer<float4, 2> output_buffer;

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry,,);

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_diffuse, prd_diffuse, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Light sources
rtBuffer<ParallelogramLight> lights;

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------

RT_PROGRAM void trace_ray()
{
	size_t2 screen = output_buffer.size(); // Screen size
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // Pixel coordinate in [-1, 1]
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Create ray from camera into scene
	Ray ray(ray_origin, ray_direction, 0, EPSILON);

	// Per radiance data
	PerRayData_diffuse prd;
	prd.seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0);//frame_number);

	// Trace geometry
	rtTrace(scene_geometry, ray, prd);

	// Set resulting color
	output_buffer[launch_index] = make_float4(prd.result, 1.0f);
}

//-----------------------------------------------------------------------------
// Lambertian surface closest-hit
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

#define FLT_MAX          3.402823466e+38F        // max value

RT_PROGRAM void diffuse()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	unsigned int seed = prd_diffuse.seed;
	for(int i = 0; i < lights.size(); ++i)
	{
		ParallelogramLight light = lights[i];
		const float3 light_center = light.corner + light.v1 * 0.5f + light.v2 * 0.5f;

		// Send 9 rays
		float d2_min = FLT_MAX; // Min distance from light to occluder
		float d2_max = 0.0f; // Max distance from light to occluder
		float d1 = length(hit_point - light_center); // Distance from light to receiver
		for(int j = 0; j < 9; j++)
		{
			// Choose random point on light
			const float z1 = rnd(seed);
			const float z2 = rnd(seed);
			const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

			float3 L = normalize(light_pos - hit_point);
			float nDl = dot(ffnormal, L);
			if(nDl > 0.0f) // Check if light is behind
			{
				float Ldist = length(light_pos - hit_point); // TODO: Maybe d1 should be average of these?

				// Cast shadow ray
				PerRayData_shadow shadow_prd;
				shadow_prd.attenuation = make_float3(1.0f);

				Ray shadow_ray(hit_point, L, 1, EPSILON, Ldist);
				rtTrace(scene_geometry, shadow_ray, shadow_prd);

				float3 light_attenuation = shadow_prd.attenuation;
				if(fmaxf(light_attenuation) > 0.0f) // If we hit the light
				{
					float3 Lc = light_attenuation * diffuse_color;
					color += Kd * nDl * Lc;

					// Apply specularity
					float3 H = normalize(L - ray.direction);
					float nDh = dot(ffnormal, H);
					if(nDh > 0)
					{
						color += Ks * Lc * pow(nDh, phong_exp);
					}
				}
				else // Else if light source was occluded
				{
					const float d2 = length(shadow_prd.hit_point - light_pos) / Ldist;

					// Store min d2
					if(d2 < d2_min)
					{
						d2_min = d2;
					}

					// Store max d2
					if(d2 > d2_max)
					{
						d2_max = d2;
					}
				}
			}
		}

		color /= 9.f;
		
		if(d2_max > 0.f)//0.01f)
			color = lerp(color, make_float3(1.0, 0.0, 0.0), 1.f - d2_max); // (600.f*2.f);

		if(d1 < 10.0f) {
			color = make_float3(1.f, 1.f, 1.f);
		}


		// Constants from the paper
		/*const float k = 3.f;
		const float alpha = 1.f;
		const float mu = 2.f;

		const float sigma = 1.f;//1.f / omega_max_L; // Standard deviation of Gaussian

		omega_max_pix = 1 / depth;
		omega_max_x = alpha * (d2_max / d1) * omega_max_pix;

		// Calculate filter width at current pixel
		beta = 1.f / k * 1.f / mu * max(sigma * ((d1 / d2_max) - 1.f), 1.f / omega_max_x);

		// Calcuate number of additional samples
		num_samples = 4 * powf(1.f + mu * (s1 / s2), 2.f) * powf(mu * 2 / s2 * sqrtf(Ap / Al) + alpha * 1.f / (1.f + s2), 2.f);*/
	}
	prd_diffuse.result = color;
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.hit_point = ray.origin + t_hit * ray.direction;
	prd_shadow.attenuation = make_float3(0);
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_diffuse.result = bg_color;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}