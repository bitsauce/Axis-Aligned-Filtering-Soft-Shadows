#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optixu/optixu_math_namespace.h>
#include <optixu/optixu_matrix_namespace.h>
#include "structs.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-1f
#define FLT_MAX 3.402823466e+38F

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

rtBuffer<float3, 2> diffuse_buffer;             // Diffuse color buffer
rtBuffer<float,  2> beta_buffer;                // Beta buffer (gaussian standard deviation)
rtBuffer<float,  2> d1_buffer;                  // Distance to light source
rtBuffer<float,  2> d2_min_buffer;              // Minimum distasnce to occluder
rtBuffer<float,  2> d2_max_buffer;              // Maximum distasnce to occluder
rtBuffer<float3, 2> geometry_hit_buffer;        // Geometry hit buffer
rtBuffer<float3, 2> geometry_normal_buffer;     // Geometry hit buffer
rtBuffer<float3, 2> ffnormal_buffer;            // For shading
rtBuffer<float,  2> object_id_buffer;           // Object id buffer
rtBuffer<float,  2> num_samples_buffer;         // Sample number buffer
rtBuffer<float2, 2> projected_distances_buffer; // Projected distances buffer (offset of screen-space gaussian)

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry, , );

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_geometry_hit, prd_geometry_hit, rtPayload, );
rtDeclareVariable(PerRayData_distances, prd_distances, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

// Light sources
rtBuffer<ParallelogramLight> lights;

// Geometry hit variables
rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );
rtDeclareVariable(float3, diffuse_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );
rtDeclareVariable(uint, object_id, , );

//--------------------------------------------------------------
// Primary ray pass
//--------------------------------------------------------------

RT_PROGRAM void trace_primary_ray()
{
	size_t2 screen = diffuse_buffer.size(); // Screen size
	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f; // Pixel coordinate in [-1, 1]
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	// Create ray from camera into scene
	Ray ray(ray_origin, ray_direction, GEOMETRY_HIT_RAY, EPSILON);

	// Per radiance data
	PerRayData_geometry_hit prd;
	prd.color = make_float3(0.f);
	prd.object_id = 0;
	prd.geometry_hit = make_float3(0.f);
	prd.geometry_normal = make_float3(0.f);
	prd.ffnormal = make_float3(0.f);

	// Trace geometry
	rtTrace(scene_geometry, ray, prd);

	// Set resulting geometry hit coordinate
	diffuse_buffer[launch_index] = prd.color;
	object_id_buffer[launch_index] = prd.object_id;
	geometry_hit_buffer[launch_index] = prd.geometry_hit;
	geometry_normal_buffer[launch_index] = prd.geometry_normal;
	ffnormal_buffer[launch_index] = prd.ffnormal;
}

RT_PROGRAM void sample_geometry_hit()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;

	prd_geometry_hit.color = diffuse_color;
	prd_geometry_hit.object_id = float(object_id);
	prd_geometry_hit.geometry_hit = hit_point;
	prd_geometry_hit.geometry_normal = world_geo_normal;
	prd_geometry_hit.ffnormal = ffnormal;
}

//--------------------------------------------------------------
// Distance sampling + adaptive sampling
//--------------------------------------------------------------

RT_PROGRAM void sample_distances_to_light(unsigned int& seed, float3 &color, ParallelogramLight light,
										  float3 ffnormal, float3 hit_point, float& d2_min, float& d2_max)
{
	// Choose random point on light
	const float z1 = rnd(seed);
	const float z2 = rnd(seed);
	const float3 light_pos = light.corner + light.v1 * z1 + light.v2 * z2;

	float3 L = normalize(light_pos - hit_point);
	float nDl = dot(ffnormal, L);
	if(nDl > 0.0f) // Check if light is behind
	{
		float Ldist = length(light_pos - hit_point);

		// Cast shadow ray
		PerRayData_shadow shadow_prd;
		shadow_prd.hit = false;

		Ray shadow_ray(hit_point, L, SHADOW_RAY, EPSILON, Ldist);
		rtTrace(scene_geometry, shadow_ray, shadow_prd);

		// If light source was occluded
		if(shadow_prd.hit)
		{
			const float d2 = length(shadow_prd.hit_point - light_pos);

			// Store min d2
			if(d2 < d2_min)
			{
				d2_min = d2;
			}

			// Store max d2
			if(d2 > d2_max)
			{
				d2_max = d2;
			}
		}
		else
		{
			const float3 Kd = make_float3(0.6f, 0.7f, 0.8f);
			color += Kd * nDl * diffuse_buffer[launch_index];
		}
	}
}

// Constants from the paper
const float k = 3.f;
const float alpha = 1.f;
const float mu = 2.f;
const float max_num_samples = 100.f;

// Standard deviation of Gaussian of the light
const float sigma = 130.f / 2.f;

RT_PROGRAM void sample_distances()
{
	size_t2 screen = geometry_hit_buffer.size();
	float3 ffnormal = ffnormal_buffer[launch_index];
	float3 hit_point = geometry_hit_buffer[launch_index];

	// Calculate projected distance per pixel
	float d = 0.f;
	if(launch_index.x > 0)        d += length(geometry_hit_buffer[make_uint2(launch_index.x - 1, launch_index.y)] - hit_point);
	if(launch_index.y > 0)        d += length(geometry_hit_buffer[make_uint2(launch_index.x, launch_index.y - 1)] - hit_point);
	if(launch_index.x < screen.x) d += length(geometry_hit_buffer[make_uint2(launch_index.x + 1, launch_index.y)] - hit_point);
	if(launch_index.y < screen.y) d += length(geometry_hit_buffer[make_uint2(launch_index.x, launch_index.y + 1)] - hit_point);
	d /= 4.f;
	const float omega_max_pix = 1.f / d;

	float3 color = make_float3(0.0f);
	unsigned int seed = tea<16>(screen.x*launch_index.y + launch_index.x, 0/*frame_number*/);
	for(int i = 0; i < lights.size(); ++i)
	{
		ParallelogramLight light = lights[i];
		const float3 light_center = light.corner + light.v1 * 0.5f + light.v2 * 0.5f;

		// Calculate distances parallel to the light source
		// (used as a offset in the gaussian blur)
		Matrix3x3 projection_matrix;
		projection_matrix.setCol(0, normalize(light.v1));
		projection_matrix.setCol(1, normalize(light.v2));
		projection_matrix.setCol(2, light.normal);

		float3 p_projected = projection_matrix * hit_point;
		projected_distances_buffer[launch_index] = make_float2(p_projected);

		// Send 9 rays
		float d2_min = FLT_MAX;  // Min distance from light to occluder
		float d2_max = -FLT_MAX; // Max distance from light to occluder
		float d1 = length(hit_point - light_center); // Distance from light to receiver
		for(int j = 0; j < 9; j++)
		{
			sample_distances_to_light(seed, color, light, ffnormal, hit_point, d2_min, d2_max);
		}

		// If this pixel was occluded (that is, d2_max > 0)
		if(d2_max > 0.f)
		{
			const float s1 = max(d1 / d2_min, 1.f) - 1.f;
			float s2 = max(d1 / d2_max, 1.f) - 1.f;
			float inv_s2 = alpha / (1.f + s2);

			// Calculate pixel area and light area
			const float Ap = 1.f / (omega_max_pix * omega_max_pix);
			const float Al = 4.f * sigma * sigma;

			// Calcuate number of additional samples
			const float num_samples = min(4.f * powf(1.f + mu * (s1 / s2), 2.f) * powf(mu * 2 / s2 * sqrtf(Ap / Al) + inv_s2, 2.f), max_num_samples);
			num_samples_buffer[launch_index] = num_samples;

			for(int j = 0; j < (int)num_samples; j++)
			{
				sample_distances_to_light(seed, color, light, ffnormal, hit_point, d2_min, d2_max);
			}

			color /= 9.f + num_samples;
		}
		else
		{
			// Set values for unoccluded pixels
			num_samples_buffer[launch_index] = 0.f;
			color /= 9.f;
			d1 = d2_min = d2_max = 0.f;
		}

		// Set sampled distances
		d1_buffer[launch_index] = d1;
		d2_min_buffer[launch_index] = d2_min;
		d2_max_buffer[launch_index] = d2_max;
	}

	// Set sampled color
	diffuse_buffer[launch_index] = color;
}

//-----------------------------------------------------------------------------
// Calculate beta
//-----------------------------------------------------------------------------

RT_PROGRAM void calculate_beta()
{
	// Calculate projected distance per pixel
	size_t2 screen = geometry_hit_buffer.size();
	float3 hit_point = geometry_hit_buffer[launch_index];
	float d = 0.f;
	if(launch_index.x > 0)        d += length(geometry_hit_buffer[make_uint2(launch_index.x - 1, launch_index.y)] - hit_point);
	if(launch_index.y > 0)        d += length(geometry_hit_buffer[make_uint2(launch_index.x, launch_index.y - 1)] - hit_point);
	if(launch_index.x < screen.x) d += length(geometry_hit_buffer[make_uint2(launch_index.x + 1, launch_index.y)] - hit_point);
	if(launch_index.y < screen.y) d += length(geometry_hit_buffer[make_uint2(launch_index.x, launch_index.y + 1)] - hit_point);
	d /= 4.f;
	const float omega_max_pix = 1.f / d;

	// Get d1, d2_max from previous pass
	float d2_max = d2_max_buffer[launch_index];
	float d1 = d1_buffer[launch_index];

	// For unocculded pixel, take the average in a 5 pixel radius
	if(d2_max == 0.f)
	{
		float sum = 0.f;
		for(int i = -5; i <= 5; i++)
		{
			for(int j = -5; j <= 5; j++)
			{
				const uint2 pos = make_uint2(launch_index.x + j, launch_index.y + i);
				if(pos.x >= screen.x || pos.y >= screen.y) continue;
				d1 += d1_buffer[pos];
				d2_max += d2_max_buffer[pos];
				sum += 1.f;
			}
		}

		// Get average
		d1 /= sum;
		d2_max /= sum;

		// Write back (for debug visualization)
		//d1_buffer[launch_index] = d1;
		//d2_max_buffer[launch_index] = d2_max;
	}

	// Make sure we can calculate beta
	if(d2_max > 0.f)
	{
		// Update s2 and inv_s2
		const float s2 = max(d1 / d2_max, 1.f) - 1.f;
		const float inv_s2 = alpha / (1.f + s2);
		const float omega_max_x = inv_s2 * omega_max_pix;

		// Calculate filter width at current pixel
		const float beta = 1.f / k * 1.f / mu * max(sigma * s2, 1.f / omega_max_x);
		beta_buffer[launch_index] = min(beta, 10.f);
	}
	else
	{
		// Pixel still unoccluded
		beta_buffer[launch_index] = 0.f;
	}
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.hit_point = ray.origin + t_hit * ray.direction;
	prd_shadow.hit = true;
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void distances_miss()
{
	prd_distances.color = bg_color;
	prd_distances.projected_distance = make_float2(0.f);
	prd_distances.d1 = prd_distances.d2_min = prd_distances.d2_max = 0.f;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	diffuse_buffer[launch_index] = bad_color;
	beta_buffer[launch_index] = 0.f;
	object_id_buffer[launch_index] = 0.f;
	num_samples_buffer[launch_index] = 0.f;
	geometry_normal_buffer[launch_index] = make_float3(0.f);
}