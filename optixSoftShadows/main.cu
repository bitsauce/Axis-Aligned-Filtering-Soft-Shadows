#include "hip/hip_runtime.h"
/* 
 * Copyright (c) 2017 NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <optix.h>
//#include "light.h"
#include "random.h"

using namespace optix;

#define EPSILON  1.e-3f

//--------------------------------------------------------------
// Per-ray data structs
//--------------------------------------------------------------

struct PerRayData_radiance
{
	float3 result;
	float  importance;
	int depth;
};

struct PerRayData_shadow
{
	float3 attenuation;
};

//--------------------------------------------------------------
// Variable declarations
//--------------------------------------------------------------

// Input pixel-coordinate
// An uint2 value (x, y) bound to internal state variable "rtLaunchIndex"
rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );

// Output buffer (final image)
// A 2-dimensional buffer of float4s
rtBuffer<float4, 2> output_buffer;

// Scene geometry objects
rtDeclareVariable(rtObject, scene_geometry,,);

// Pinhole camera variables
rtDeclareVariable(float3, eye, , );
rtDeclareVariable(float3, U,   , );
rtDeclareVariable(float3, V,   , );
rtDeclareVariable(float3, W,   , );

rtDeclareVariable(float3, geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3, shading_normal, attribute shading_normal, );

rtDeclareVariable(Ray, ray, rtCurrentRay, );
rtDeclareVariable(PerRayData_radiance, prd_radiance, rtPayload, );
rtDeclareVariable(PerRayData_shadow, prd_shadow, rtPayload, );

//rtBuffer<ParallelogramLight> lights;

//--------------------------------------------------------------
// Main ray program
//--------------------------------------------------------------

RT_PROGRAM void trace_ray()
{
	size_t2 screen = output_buffer.size();

	float2 d = make_float2(launch_index) / make_float2(screen) * 2.f - 1.f;
	float3 ray_origin = eye;
	float3 ray_direction = normalize(d.x*U + d.y*V + W);

	Ray ray(ray_origin, ray_direction, 0, EPSILON);

	PerRayData_radiance prd;
	prd.importance = 1.f;
	prd.depth = 0;

	rtTrace(scene_geometry, ray, prd);

	output_buffer[launch_index] = make_float4(prd.result, 1.0f);
}

//-----------------------------------------------------------------------------
// Lambertian surface closest-hit
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, Ka, , );
rtDeclareVariable(float3, Ks, , );
rtDeclareVariable(float, phong_exp, , );
rtDeclareVariable(float3, Kd, , );
rtDeclareVariable(float3, ambient_light_color, , );
rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

struct BasicLight
{
#if defined(__cplusplus)
	typedef optix::float3 float3;
#endif
	float3 pos;
	float3 color;
	int    casts_shadow;
	int    padding;
};

rtBuffer<BasicLight> lights;


RT_PROGRAM void diffuse()
{
	float3 world_geo_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 color = Ka * ambient_light_color;

	float3 hit_point = ray.origin + t_hit * ray.direction;

	for(int i = 0; i < lights.size(); ++i) {
		BasicLight light = lights[i];
		float3 L = normalize(light.pos - hit_point);
		float nDl = dot(ffnormal, L);

		if(nDl > 0.0f) {
			// cast shadow ray
			PerRayData_shadow shadow_prd;
			shadow_prd.attenuation = make_float3(1.0f);
			float Ldist = length(light.pos - hit_point);
			Ray shadow_ray(hit_point, L, 1, EPSILON, Ldist);
			rtTrace(scene_geometry, shadow_ray, shadow_prd);
			float3 light_attenuation = shadow_prd.attenuation;

			if(fmaxf(light_attenuation) > 0.0f) {
				float3 Lc = light.color * light_attenuation;
				color += Kd * nDl * Lc;

				float3 H = normalize(L - ray.direction);
				float nDh = dot(ffnormal, H);
				if(nDh > 0)
					color += Ks * Lc * pow(nDh, phong_exp);
			}

		}
	}
	prd_radiance.result = color;
}

//-----------------------------------------------------------------------------
// Shadow any-hit
//-----------------------------------------------------------------------------

RT_PROGRAM void shadow()
{
	prd_shadow.attenuation = make_float3(0);
	rtTerminateRay();
}

//--------------------------------------------------------------
// Miss program
//--------------------------------------------------------------

rtDeclareVariable(float3, bg_color,,);

RT_PROGRAM void miss()
{
	prd_radiance.result = bg_color;
}

//--------------------------------------------------------------
// Exception
//--------------------------------------------------------------

rtDeclareVariable(float3, bad_color,,);

RT_PROGRAM void exception()
{
	output_buffer[launch_index] = make_float4(bad_color, 1.0f);
}