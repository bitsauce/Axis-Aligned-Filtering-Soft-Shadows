#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Normalizes the input buffer by dividing its components by
// the max value
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float, 2> normalize_buffer;
rtBuffer<float3, 2> heatmap_buffer;
rtDeclareVariable(float, max_value, , );

RT_PROGRAM void normalize()
{
	normalize_buffer[launch_index] /= max_value;

	float greyValue = normalize_buffer[launch_index];
	float3 heat = make_float3(0.f, 0.f, 0.f);

	heat.x = smoothstep(0.5f, 0.8f, greyValue);
	if (greyValue >= 0.90f) {
		heat.x *= (1.1f - greyValue) * 5.0f;
	}
	if (greyValue > 0.7f) {
		heat.y = smoothstep(1.0f, 0.7f, greyValue);
	}
	else {
		heat.y = smoothstep(0.0f, 0.7f, greyValue);
	}
	heat.z = smoothstep(1.0f, 0.0f, greyValue);
	if (greyValue <= 0.3f) {
		heat.z *= greyValue / 0.3f;
	}

	heatmap_buffer[launch_index] = heat;
}
