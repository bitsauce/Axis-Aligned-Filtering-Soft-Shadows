#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>

using namespace optix;

//--------------------------------------------------------------
// Normalizes the input buffer by dividing its components by
// the max value
//--------------------------------------------------------------

rtDeclareVariable(uint2, launch_index, rtLaunchIndex, );
rtBuffer<float, 2> normalize_buffer;
rtDeclareVariable(float, max_value, , );

RT_PROGRAM void normalize()
{
	normalize_buffer[launch_index] /= max_value;
}
